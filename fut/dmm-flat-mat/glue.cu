#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "hipblas.h"
#include <pthread.h>

#define SIZE 256

#define ENABLE_TIMER_TICKS false

// ==========================================================================
// timer stuff

struct my_timer_t {
  const char *name;
  struct timespec start;
  struct timespec most_recent_tick;
};

static void timespec_subtract(struct timespec *x, struct timespec *y) {
  if (x->tv_nsec < y->tv_nsec) {
    x->tv_sec -= 1;
    x->tv_nsec += 1000000000L;
  }
  x->tv_sec -= y->tv_sec;
  x->tv_nsec -= y->tv_nsec;
}

static void report_elapsed(
  const char *name, 
  const char *msg, 
  struct timespec *x, 
  struct timespec *y)
{
  struct timespec diff = *x;
  timespec_subtract(&diff, y);
  double secs = (double)diff.tv_sec + ((double)diff.tv_nsec / 1000000000.0);
  printf("tick: %s: %s: elapsed: %lf\n", name, msg, secs);
}

void timer_begin(struct my_timer_t *t, const char *name) {
  t->name = name;
  clock_gettime(CLOCK_MONOTONIC, &(t->start));
  t->most_recent_tick = t->start;
}

void timer_report_tick(struct my_timer_t *t, const char *msg) {
  struct timespec prev = t->most_recent_tick;
  clock_gettime(CLOCK_MONOTONIC, &(t->most_recent_tick));
  if (ENABLE_TIMER_TICKS) {
    report_elapsed(t->name, msg, &(t->most_recent_tick), &prev);
  }
}


// ==========================================================================
// dMM boilerplate


/* TODO: inputs and outputs for leaf DMM, dimension info, etc. */
struct dMMPackage {
  // struct futStuff *futStuff;  /* won't need this */

  /* need to be specialized for DMM */
  float * a;
  float * b;
  float * output;
  uint64_t inputLen;

  /* these should stay */
  bool finished;
  pthread_t friends;
};

/* TODO: call cublas */
void* asyncdMMFunc(void* rawArg) {
  struct my_timer_t t;
  timer_begin(&t, "asyncdMMFunc");

  struct dMMPackage *pack = (struct dMMPackage *)rawArg;

  float *device_a;
  float *device_b;
  float *device_output;

  uint64_t n = pack->inputLen;
  uint64_t bytes = n*n*sizeof(float);

  hipMalloc(&device_a, bytes);
  hipMemcpy(device_a, pack->a, bytes, hipMemcpyHostToDevice);

  hipMalloc(&device_b, bytes);
  hipMemcpy(device_b, pack->b, bytes, hipMemcpyHostToDevice);
  
  hipMalloc(&(device_output), bytes);
  hipMemcpy(device_output, pack->output, bytes, hipMemcpyHostToDevice);
  // timer_report_tick(&t, "--- memcpy to gpu");

  float alpha = 1.0;
  float beta = 1.0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);  
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, device_a, n, device_b, n, &beta, device_output, n);
  hipblasDestroy(handle);
  // timer_report_tick(&t, "      cublasSgemm");

  hipMemcpy(pack->output, device_output, bytes, hipMemcpyDeviceToHost);
  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_output);
  // timer_report_tick(&t, "  memcpy from gpu");

  pack->finished = true; /* VERY IMPORTANT! */
  return NULL;
}


/* TODO: build the package, but otherwise shouldn't need to change much. 
 *
 * (NOTE: futhark_new_... is essentially a memcpy, these need to be replaced
 *  with stuff for cublas)
 */
extern "C" struct dMMPackage * 
dMMSpawn(
  float * a,
  float * b,
  float * output,
  int64_t inputLen)
{
  // struct futhark_context *ctx = futStuff->ctx;
  struct dMMPackage *pack = (dMMPackage*)malloc(sizeof(struct dMMPackage));
  // pack->futStuff = futStuff;
  // pack->a = futhark_new_u8_1d(ctx, a, inputLen);

  pack->a = a;
  pack->b = b;
  pack->output = output;
  pack->inputLen = inputLen;
  pack->finished = false;

  if (0 != pthread_create(&(pack->friends), NULL, &asyncdMMFunc, pack)) {
    printf("ERROR: glue.c: futdMMSpawn: pthread_create failed\n");
    exit(1);
  }

  return pack;
}

extern "C" uint8_t dMMPoll(struct dMMPackage *pack) {
  return pack->finished ? 1 : 0;
}


/* TODO: memcpy from GPU back to pack->output
 *
 * (NOTE: futhark_values is equivalent of this memcpy. needs to be replaced) */
extern "C" void dMMFinish(
  struct dMMPackage * pack)
{
  if (0 != pthread_join(pack->friends, NULL)) {
    printf("ERROR: glue.c: pthread_join failed\n");
    exit(1);
  }

  free(pack);
}

// ==========================================================================

extern "C"
void * memcpyFloatsToGpu(float *data, int64_t len) {
  struct my_timer_t t;
  timer_begin(&t, "memcpyFloatsToGpu");

  float *p;
  hipMalloc(&p, len*sizeof(float));
  hipMemcpyAsync(p, data, len*sizeof(float), hipMemcpyHostToDevice);

  timer_report_tick(&t, "done");
  return p;
}

extern "C"
void synchronizeGpu() {
  hipDeviceSynchronize();
}

extern "C"
void freeFloatsOnGpu(void *devicePtr) {
  hipFree(devicePtr);
}


// ==========================================================================


// copy into dst[0..n*n)
__global__
void copy_square_block(
  float *dst,
  uint64_t n,
  float *src,
  uint64_t top,
  uint64_t left,
  uint64_t rowskip)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  int total = n*n;
  for (int i = tid; i < total; i += stride) {
    int row = i/n;
    int col = i%n;
    int srcIdx = (top + row) * rowskip + left + col;
    dst[i] = src[srcIdx];
  }
}


// ==========================================================================


struct fancy_dmm_package {
  float * a;  // on device
  int64_t aTop;
  int64_t aLeft;
  int64_t aRowskip;
  float * b;  // on device
  int64_t bTop;
  int64_t bLeft;
  int64_t bRowskip;
  float * c;  // on host
  int64_t cTop;
  int64_t cLeft;
  int64_t cRowskip;
  int64_t n;

  /* these should stay */
  bool finished;
  pthread_t friends;
};



extern "C"
void* fancy_dmm_func(void* rawArg) {
  struct my_timer_t t;
  timer_begin(&t, "fancy_dmm_func");

  struct fancy_dmm_package *pack = (struct fancy_dmm_package *)rawArg;

  uint64_t n = pack->n;
  uint64_t rowbytes = n*sizeof(float);
  uint64_t bytes = n*rowbytes;


  float *device_a;
  float *device_b;
  float *device_c;
  hipMalloc(&device_a, bytes);
  hipMalloc(&device_b, bytes);
  hipMalloc(&device_c, bytes);

  int GRID = ((n*n)+(SIZE-1))/SIZE;
  if(GRID == 0) {
    GRID = 1;
  }
  copy_square_block<<<GRID, SIZE>>>(device_a, n, pack->a, pack->aTop, pack->aLeft, pack->aRowskip);
  // cudaDeviceSynchronize();

  copy_square_block<<<GRID, SIZE>>>(device_b, n, pack->b, pack->bTop, pack->bLeft, pack->bRowskip);
  hipDeviceSynchronize();

  timer_report_tick(&t, "--- memcpy to gpu");

  float alpha = 1.0;
  float beta = 0.0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);  
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, device_a, n, device_b, n, &beta, device_c, n);
  hipblasDestroy(handle);
  timer_report_tick(&t, "      hipblasSgemm");


  hipMemcpy(pack->c, device_c, bytes, hipMemcpyDeviceToHost);

  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_c);
  timer_report_tick(&t, "  memcpy from gpu");

  pack->finished = true; /* VERY IMPORTANT! */
  return NULL;
}


extern "C" struct fancy_dmm_package * 
fancy_dmm_spawn(
  float * a,     // on device
  int64_t aTop,
  int64_t aLeft,
  int64_t aRowskip,
  float * b,     // on device
  int64_t bTop,
  int64_t bLeft,
  int64_t bRowskip,
  float * c,     // on host
  int64_t cTop,
  int64_t cLeft,
  int64_t cRowskip,
  int64_t n)
{
  struct fancy_dmm_package *pack = (fancy_dmm_package*)malloc(sizeof(struct fancy_dmm_package));

  pack->a = a;
  pack->aTop = aTop;
  pack->aLeft = aLeft;
  pack->aRowskip = aRowskip;

  pack->b = b;
  pack->bTop = bTop;
  pack->bLeft = bLeft;
  pack->bRowskip = bRowskip;

  pack->c = c;
  pack->cTop = cTop;
  pack->cLeft = cLeft;
  pack->cRowskip = cRowskip;

  pack->n = n;

  pack->finished = false;
  if (0 != pthread_create(&(pack->friends), NULL, &fancy_dmm_func, pack)) {
    printf("ERROR: glue.c: futdMMSpawn: pthread_create failed\n");
    exit(1);
  }

  return pack;
}


extern "C" uint8_t fancy_dmm_poll(struct fancy_dmm_package *pack) {
  return pack->finished ? 1 : 0;
}


/* TODO: memcpy from GPU back to pack->output
 *
 * (NOTE: futhark_values is equivalent of this memcpy. needs to be replaced) */
extern "C" void fancy_dmm_finish(
  struct fancy_dmm_package * pack)
{
  if (0 != pthread_join(pack->friends, NULL)) {
    printf("ERROR: glue.c: pthread_join failed\n");
    exit(1);
  }

  free(pack);
}


// ===========================================================================


struct fancy_two_dmm_package {
  float * a;  // on device
  int64_t aTop1;
  int64_t aLeft1;
  int64_t aTop2;
  int64_t aLeft2;
  int64_t aRowskip;
  float * b;  // on device
  int64_t bTop1;
  int64_t bLeft1;
  int64_t bTop2;
  int64_t bLeft2;
  int64_t bRowskip;
  float * c;  // on host
  int64_t cTop;
  int64_t cLeft;
  int64_t cRowskip;
  int64_t n;

  /* these should stay */
  bool finished;
  pthread_t friends;
};



extern "C"
void* fancy_two_dmm_func(void* rawArg) {
  struct my_timer_t t;
  timer_begin(&t, "fancy_two_dmm_func");

  struct fancy_two_dmm_package *pack = (struct fancy_two_dmm_package *)rawArg;

  uint64_t n = pack->n;
  uint64_t rowbytes = n*sizeof(float);
  uint64_t bytes = n*rowbytes;


  float *device_c;
  hipMalloc(&device_c, bytes);
  // for (int64_t j = 0; j < n; j++) {
  //   float *host_start = pack->c + (pack->cTop + j) * pack->cRowskip + pack->cLeft;
  //   cudaMemcpyAsync(device_c + j*n, host_start, rowbytes, cudaMemcpyHostToDevice);
  // }

  // cudaDeviceSynchronize();
  // timer_report_tick(&t, "----- memcpy C to gpu");

  float *tmp_a;
  float *tmp_b;
  hipMalloc(&tmp_a, bytes);
  hipMalloc(&tmp_b, bytes);


  int GRID = ((n*n)+(SIZE-1))/SIZE;
  if(GRID == 0) {
    GRID = 1;
  }
  copy_square_block<<<GRID, SIZE>>>(tmp_a, n, pack->a, pack->aTop1, pack->aLeft1, pack->aRowskip);
  copy_square_block<<<GRID, SIZE>>>(tmp_b, n, pack->b, pack->bTop1, pack->bLeft1, pack->bRowskip);
  hipDeviceSynchronize();
  timer_report_tick(&t, "- memcpy A1,B1 on gpu");

  float alpha = 1.0;
  float beta = 0.0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);  
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, tmp_a, n, tmp_b, n, &beta, device_c, n);
  hipblasDestroy(handle);
  timer_report_tick(&t, "   cublasSgemm(A1,B1)");
  

  copy_square_block<<<GRID, SIZE>>>(tmp_a, n, pack->a, pack->aTop2, pack->aLeft2, pack->aRowskip);
  copy_square_block<<<GRID, SIZE>>>(tmp_b, n, pack->b, pack->bTop2, pack->bLeft2, pack->bRowskip);
  hipDeviceSynchronize();
  timer_report_tick(&t, "  memcpy A2,B2 on gpu");


  hipblasCreate(&handle);  
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, tmp_a, n, tmp_b, n, &beta, device_c, n);
  hipblasDestroy(handle);
  timer_report_tick(&t, "   cublasSgemm(A2,B2)");

  // for (int64_t j = 0; j < n; j++) {
  //   float *host_start = pack->c + (pack->cTop + j) * pack->cRowskip + pack->cLeft;
  //   cudaMemcpyAsync(host_start, device_c + j*n, rowbytes, cudaMemcpyDeviceToHost);
  // }
  // cudaDeviceSynchronize();

  hipMemcpy(pack->c, device_c, bytes, hipMemcpyDeviceToHost);
  hipFree(tmp_a);
  hipFree(tmp_b);
  hipFree(device_c);
  timer_report_tick(&t, "    memcpy C from gpu");

  pack->finished = true; /* VERY IMPORTANT! */
  return NULL;
}


extern "C" struct fancy_two_dmm_package * 
fancy_two_dmm_spawn(
  float * a,     // on device
  int64_t aTop1,
  int64_t aLeft1,
  int64_t aTop2,
  int64_t aLeft2,
  int64_t aRowskip,
  float * b,     // on device
  int64_t bTop1,
  int64_t bLeft1,
  int64_t bTop2,
  int64_t bLeft2,
  int64_t bRowskip,
  float * c,     // on host
  int64_t cTop,
  int64_t cLeft,
  int64_t cRowskip,
  int64_t n)
{
  struct fancy_two_dmm_package *pack = (fancy_two_dmm_package*)malloc(sizeof(struct fancy_two_dmm_package));

  pack->a = a;
  pack->aTop1 = aTop1;
  pack->aLeft1 = aLeft1;
  pack->aTop2 = aTop2;
  pack->aLeft2 = aLeft2;
  pack->aRowskip = aRowskip;

  pack->b = b;
  pack->bTop1 = bTop1;
  pack->bLeft1 = bLeft1;
  pack->bTop2 = bTop2;
  pack->bLeft2 = bLeft2;
  pack->bRowskip = bRowskip;

  pack->c = c;
  pack->cTop = cTop;
  pack->cLeft = cLeft;
  pack->cRowskip = cRowskip;

  pack->n = n;

  pack->finished = false;
  if (0 != pthread_create(&(pack->friends), NULL, &fancy_two_dmm_func, pack)) {
    printf("ERROR: glue.c: futdMMSpawn: pthread_create failed\n");
    exit(1);
  }

  return pack;
}


extern "C" uint8_t fancy_two_dmm_poll(struct fancy_two_dmm_package *pack) {
  return pack->finished ? 1 : 0;
}


extern "C" void fancy_two_dmm_finish(
  struct fancy_two_dmm_package * pack)
{
  if (0 != pthread_join(pack->friends, NULL)) {
    printf("ERROR: glue.c: pthread_join failed\n");
    exit(1);
  }

  free(pack);
}